
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

#define THREADS_PER_BLOCK 1024


void vec_mul(double *z, const double* v, const double* w, long N) {
  double prod = 0;
  #pragma omp parallel for schedule(static) reduction(+:prod)
  for (long i = 0; i < N; i++) {
  	prod += v[i] * w[i];
  }
  z[0] = prod;
}

void matrix_vec_mul(double* z, double* A, double* x,long M, long N) {
  for (long i = 0; i < M; i++) {
    vec_mul(z+i, A+i*N,x,N);
  }
}



__global__ void matrix_vec_mul_kernel(double* z, double* A, double* x,long M, long N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < M) {
    double sum = 0;
    for (long i = 0; i < N; i++) sum += A[idx*N+i]*x[i];
    z[idx] = sum;
  }
}


int main() {
  long M = (1UL<<13);
  long N = (1UL<<13);

  double * A, *A_d, *x, *x_d, *z, *z_d, *z_ref;
  A = (double*) malloc(M*N * sizeof(double));
  x = (double*) malloc(N * sizeof(double));
  z = (double*) malloc(M * sizeof(double));
  z_ref = (double*) malloc(M * sizeof(double));

  checkCuda( hipMalloc(&A_d, M*N * sizeof(double)));
  checkCuda( hipMalloc(&x_d, N * sizeof(double)));
  checkCuda( hipMalloc(&z_d, M * sizeof(double)));

  #pragma omp parallel for schedule(static)
  for (long i = 0; i < M; i++) {
      z[i] = 0;
      z_ref[i] = 0;
  }
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
      x[i] = i+2;
  }
  #pragma omp parallel for schedule(static) collapse(2)
  for (long i = 0; i < M; i++) {
      for (long j = 0; j < N; j++) { 
        A[i*N+j]=j+1+(M-i);
      }
  }

  // cudaDeviceProp prop;
  // checkCuda( cudaGetDeviceProperties(&prop, 0));
  // printf("\nDevice : %s\n", prop.name);
  // printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
  // printf("Compute capability: %d.%d\n", prop.major, prop.minor);
  // printf("\nM %ld, N %ld\n", M, N);

  double tt = omp_get_wtime();
  matrix_vec_mul(z_ref, A, x, M,N);
  tt = omp_get_wtime()-tt;
  printf("CPU time %f, Bandwidth = %f GB/s\n",tt, 3*M*N*sizeof(double) / tt/1e9);


  tt = omp_get_wtime();
  checkCuda( hipMemcpy(A_d, A, M*N*sizeof(double), hipMemcpyHostToDevice));
  checkCuda( hipMemcpy(x_d, x, N*sizeof(double), hipMemcpyHostToDevice));
  checkCuda( hipMemcpy(z_d, z, M*sizeof(double), hipMemcpyHostToDevice));

  double tt1 = omp_get_wtime();
  matrix_vec_mul_kernel<<<M/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(z_d, A_d, x_d, M,N);
  hipDeviceSynchronize();
  tt1 = omp_get_wtime() - tt1;

  checkCuda( hipMemcpy(z, z_d, M*sizeof(double), hipMemcpyDeviceToHost));
  tt = omp_get_wtime()-tt;
  printf("GPU time %f, Bandwidth = %f GB/s\n", tt, 3*M*N*sizeof(double) / tt/1e9);
  printf("GPU inner time %f\n", tt1);

  double err = 0;
  for (long i = 0; i < M; i++) err += fabs(z[i]-z_ref[i]);
  printf("Error = %f\n", err);
  


  free(A);
  free(x);
  free(z);
  free(z_ref);
  checkCuda( hipFree(A_d));
  checkCuda( hipFree(x_d));
  checkCuda( hipFree(z_d));
}